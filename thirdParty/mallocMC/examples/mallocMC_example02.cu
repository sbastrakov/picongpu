#include "hip/hip_runtime.h"
/*
  mallocMC: Memory Allocator for Many Core Architectures.
  https://www.hzdr.de/crp

  Copyright 2014 Institute of Radiation Physics,
                 Helmholtz-Zentrum Dresden - Rossendorf

  Author(s):  Carlchristian Eckert - c.eckert ( at ) hzdr.de

  Permission is hereby granted, free of charge, to any person obtaining a copy
  of this software and associated documentation files (the "Software"), to deal
  in the Software without restriction, including without limitation the rights
  to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
  copies of the Software, and to permit persons to whom the Software is
  furnished to do so, subject to the following conditions:

  The above copyright notice and this permission notice shall be included in
  all copies or substantial portions of the Software.

  THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
  IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
  FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
  AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
  LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
  OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
  THE SOFTWARE.
*/

#include <iostream>
#include <cassert>
#include <vector>
#include <numeric>

#include <hip/hip_runtime.h>
#include <boost/mp11/integral.hpp>

///////////////////////////////////////////////////////////////////////////////
// includes for mallocMC
///////////////////////////////////////////////////////////////////////////////
// basic files for mallocMC
#include "src/include/mallocMC/mallocMC_hostclass.hpp"

// Load all available policies for mallocMC
#include "src/include/mallocMC/CreationPolicies.hpp"
#include "src/include/mallocMC/DistributionPolicies.hpp"
#include "src/include/mallocMC/OOMPolicies.hpp"
#include "src/include/mallocMC/ReservePoolPolicies.hpp"
#include "src/include/mallocMC/AlignmentPolicies.hpp"

///////////////////////////////////////////////////////////////////////////////
// Configuration for mallocMC
///////////////////////////////////////////////////////////////////////////////

// configurate the CreationPolicy "Scatter"
struct ScatterConfig{
    typedef boost::mp11::mp_int<4096> pagesize;
    typedef boost::mp11::mp_int<8>    accessblocks;
    typedef boost::mp11::mp_int<16>   regionsize;
    typedef boost::mp11::mp_int<2>    wastefactor;
    typedef boost::mp11::mp_false     resetfreedpages;
};

struct ScatterHashParams{
    typedef boost::mp11::mp_int<38183> hashingK;
    typedef boost::mp11::mp_int<17497> hashingDistMP;
    typedef boost::mp11::mp_int<1>     hashingDistWP;
    typedef boost::mp11::mp_int<1>     hashingDistWPRel;
};

// configure the DistributionPolicy "XMallocSIMD"
struct DistributionConfig{
  typedef ScatterConfig::pagesize pagesize;
};

// configure the AlignmentPolicy "Shrink"
struct AlignmentConfig{
  typedef boost::mp11::mp_int<16> dataAlignment;
};

// Define a new mMCator and call it ScatterAllocator
// which resembles the behaviour of ScatterAlloc
typedef mallocMC::Allocator<
  mallocMC::CreationPolicies::Scatter<ScatterConfig,ScatterHashParams>,
  mallocMC::DistributionPolicies::XMallocSIMD<DistributionConfig>,
  mallocMC::OOMPolicies::ReturnNull,
  mallocMC::ReservePoolPolicies::SimpleCudaMalloc,
  mallocMC::AlignmentPolicies::Shrink<AlignmentConfig>
  > ScatterAllocator;


///////////////////////////////////////////////////////////////////////////////
// End of mallocMC configuration
///////////////////////////////////////////////////////////////////////////////


void run();

int main()
{
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    if( deviceProp.major < int(2) ) {
        std::cerr << "Error: Compute Capability >= 2.0 required. (is ";
        std::cerr << deviceProp.major << "."<< deviceProp.minor << ")" << std::endl;
        return 1;
    }

    hipSetDevice(0);
    run();
    hipDeviceReset();

    return 0;
}


__device__ int** arA;
__device__ int** arB;
__device__ int** arC;


__global__ void createArrayPointers(int x, int y, ScatterAllocator::AllocatorHandle  mMC){
    arA = (int**) mMC.malloc(sizeof(int*) * x*y);
    arB = (int**) mMC.malloc(sizeof(int*) * x*y);
    arC = (int**) mMC.malloc(sizeof(int*) * x*y);
}


__global__ void fillArrays(int length, int* d, ScatterAllocator::AllocatorHandle mMC){
    int id = threadIdx.x + blockIdx.x*blockDim.x;

    arA[id] = (int*) mMC.malloc(sizeof(int)*length);
    arB[id] = (int*) mMC.malloc(sizeof(int)*length);
    arC[id] = (int*) mMC.malloc(sizeof(int)*length);

    for(int i=0 ; i<length; ++i){
        arA[id][i] = id*length+i;
        arB[id][i] = id*length+i;
    }
}


__global__ void addArrays(int length, int* d){
    int id = threadIdx.x + blockIdx.x*blockDim.x;

    d[id] = 0;
    for(int i=0 ; i<length; ++i){
        arC[id][i] = arA[id][i] + arB[id][i];
        d[id] += arC[id][i];
    }
}


__global__ void freeArrays(ScatterAllocator::AllocatorHandle mMC){
    int id = threadIdx.x + blockIdx.x*blockDim.x;
    mMC.free(arA[id]);
    mMC.free(arB[id]);
    mMC.free(arC[id]);
}


__global__ void freeArrayPointers(ScatterAllocator::AllocatorHandle mMC){
    mMC.free(arA);
    mMC.free(arB);
    mMC.free(arC);
}


void run()
{
    size_t block = 32;
    size_t grid = 32;
    int length = 100;
    assert((unsigned)length <= block*grid); //necessary for used algorithm

    //init the heap
    std::cerr << "initHeap...";
    ScatterAllocator mMC(1U*1024U*1024U*1024U); //1GB for device-side malloc
    std::cerr << "done" << std::endl;

    // device-side pointers
    int*  d;
    hipMalloc((void**) &d, sizeof(int)*block*grid);

    // host-side pointers
    std::vector<int> array_sums(block*grid,0);

    // create arrays of arrays on the device
    createArrayPointers<<<1,1>>>(grid, block, mMC );

    // fill 2 of them all with ascending values
    fillArrays<<<grid,block>>>(length, d, mMC );

    // add the 2 arrays (vector addition within each thread)
    // and do a thread-wise reduce to d
    addArrays<<<grid,block>>>(length, d);

    hipMemcpy(&array_sums[0], d, sizeof(int)*block*grid, hipMemcpyDeviceToHost);

    int sum = std::accumulate(array_sums.begin(), array_sums.end(), 0);
    std::cout << "The sum of the arrays on GPU is " << sum << std::endl;

    int n = block*grid*length;
    int gaussian = n*(n-1);
    std::cout << "The gaussian sum as comparison: " << gaussian << std::endl;

    // checking the free memory of the allocator
    if(mallocMC::Traits<ScatterAllocator>::providesAvailableSlots){
        std::cout << "there are ";
        std::cout << mMC.getAvailableSlots(1024U*1024U);
        std::cout << " Slots of size 1MB available" << std::endl;
    }

    freeArrays<<<grid, block>>>( mMC );
    freeArrayPointers<<<1, 1>>>( mMC );
    hipFree(d);

}
